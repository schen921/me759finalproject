#include <iostream>
#include <cstdlib>
#include "network.h"

int main(int argc, char *argv[]) {

    size_t n = atoi(argv[1]);
    int mode = atoi(argv[2]);

    // setting
    int input_length = 4;
    int input_channel = 1;
    int kernel_length = 2;
    int output_channel = 1;

    hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float ms;

    img *input, *output;
    ker *kernel;

    hipMallocManaged((void **)&input, sizeof(img));
    hipMallocManaged((void **)&kernel, sizeof(ker));
    hipMallocManaged((void **)&output, sizeof(img));

    int padding = 0;
    int stride = 1;

    input->length = input_length;
    input->channel = input_channel;

    kernel->length = kernel_length;
    kernel->bias = 0;

    output->length = ((input->length + 2 * padding) - kernel->length)/stride + 1;
    output->channel = output_channel;

    kernel->channel_out = output->channel;
    kernel->channel_in = input->channel;

    hipMallocManaged((void **)&input->content,
            n * input->length * input->length * input->channel * sizeof(float));
    hipMallocManaged((void **)&output->content,
            n * output->length * output->length * output->channel *sizeof(float));

    hipMallocManaged((void **)&kernel->content,
            kernel->length * kernel->length * kernel->channel_out * kernel->channel_in * sizeof(float));
    hipMallocManaged((void **)&kernel->bias, kernel->channel_out*sizeof(float));

    // modify input image
    for (int i = 0; i<n * input->length * input->length * input->channel; i++) {
        *(input->content + i) = i;
    }

    // modify kernel
    for (int i = 0; i<kernel->length*kernel->length*kernel->channel_in*kernel->channel_out; i++) {
        *(kernel->content + i) = i;
    }

    for (int i = 0; i<kernel->channel_out; i++) {
        *(kernel->bias + i) = 0;
    }

    hipEventRecord(start);
    if (mode == 0) {
        convolution(input, output, kernel, padding, stride, n);
    } else {
        accelerated_convolution(input, output, kernel, padding, stride, n);
    }
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);

    printf("%.2f\n", ms);

    hipFree(input->content);
    hipFree(kernel->content);
    hipFree(kernel->bias);
    hipFree(output->content);

	hipFree(input);
	hipFree(kernel);
	hipFree(output);
}